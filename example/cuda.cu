#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "co_context/lazy_io.hpp"

#define CHECK(callee)                                            \
    do {                                                         \
        const hipError_t err = callee;                          \
        if (err == hipSuccess) break;                           \
        printf("CUDA error at %s(%d)\n", __FILE__, __LINE__);    \
        printf("    Function:   %s\n", __FUNCTION__);            \
        printf("    Error code: %d\n", err);                     \
        printf("    Error hint: %s\n", hipGetErrorString(err)); \
        exit(1);                                                 \
    } while (0)

constexpr unsigned int FULL_MASK = 0xffffffff;

void __global__ reduce_cp(const double *d_x, double *d_y, const int N) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ double s_y[];

    double y = 0.0;
    const int stride = blockDim.x * gridDim.x; // 以网格大小为跨度
    for (int n = bid * blockDim.x + tid; n < N; n += stride) {
        y += d_x[n]; // 确保一个网格能覆盖所有数据
    }
    s_y[tid] = y;
    __syncthreads();

    // 线程块内，跨线程束折半归约
    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1) {
        if (tid < offset) {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    y = s_y[tid];

    for (int offset = 16; offset > 0; offset >>= 1) {
        y += __shfl_down_sync(FULL_MASK, y, offset);
    }

    if (tid == 0) {
        d_y[bid] = y; // 返回线程块结果
    }
}

constexpr int N = 1e8;
constexpr int BLOCK_LEN = 128;
constexpr int GRID_SIZE = 10240;
__device__ double d_input[N];
__device__ double d_output[GRID_SIZE];

co_context::task<double> reduce(const double *d_x) {
    double *d_y;
    CHECK(hipGetSymbolAddress((void **)&d_y, d_output));
    constexpr int shared_size = sizeof(double) * BLOCK_LEN;
    reduce_cp<<<GRID_SIZE, BLOCK_LEN, shared_size>>>(d_x, d_y, N);
    reduce_cp<<<1, 1024, sizeof(double) * 1024>>>(d_y, d_y, GRID_SIZE);

    double h_y[1] = {0};
    CHECK(hipMemcpy(h_y, d_y, sizeof(double), hipMemcpyDeviceToHost));
    // CHECK(hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_output), sizeof(double)));

    co_return h_y[0];
}

int main() {
    static double input[N];
    std::fill_n(input, N, 1.23f);
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_input), input, N * sizeof(double)));
    double *d_x;
    CHECK(hipGetSymbolAddress((void **)&d_x, d_input));
    printf("%f\n", reduce(d_x));
    return 0;
}
